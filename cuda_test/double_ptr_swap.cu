#include <iostream>
#include <algorithm>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cassert>

//! Get the block id
__device__ int block_idx(int grid_dim) {
  int block_id = blockIdx.x + (grid_dim == 2 ? 1 : 0) * blockIdx.y * gridDim.x +
                 (grid_dim == 3 ? 1 : 0) * blockIdx.z * gridDim.x * gridDim.y;
  return block_id;
}

//! Get the global thread id
__device__ int thread_idx(int grid_dim, int block_dim) {
  // thread id inside a block
  unsigned long int threadIdInBlock =
      threadIdx.x + (block_dim == 2 ? 1 : 0) * threadIdx.y * blockDim.x +
      (block_dim == 3 ? 1 : 0) * threadIdx.z * blockDim.x * blockDim.z;
  // block id
  unsigned long int block_id = block_idx(grid_dim);
  // block size
  unsigned long int threadsPerblock = blockDim.x *
                                      (block_dim == 2 ? blockDim.y : 1) *
                                      (block_dim == 3 ? blockDim.z : 1);
  unsigned long int thread_id = block_id * threadsPerblock + threadIdInBlock;
  return thread_id;
}

//! Get the transposed matrix
__global__ void gpu_array_swap(int* ptr_gpu, int nbrows, int nbcols,
                               int grid_dim, int block_dim) {
  int thread_id = thread_idx(grid_dim, block_dim);
  // check if the array is correct.
  printf("%d %d\n", ptr_gpu[thread_id], thread_id);

  // pass the matrix to shared memory
  extern __shared__ int sdata[];
  sdata[thread_id] = ptr_gpu[thread_id];

  __syncthreads();
  int row = floorf((thread_id + 1) / nbcols);
  int col = thread_id + 1 - row * nbcols;
  int swap_thread_id = col * nbcols + row - 1;
  ptr_gpu[thread_id] = sdata[swap_thread_id];
  printf("%d %d\n", ptr_gpu[thread_id], thread_id);
}

void print_array(int** array, int nbrows, int nbcols) {
  for (int i = 0; i < nbrows; ++i) {
    for (int j = 0; j < nbcols; ++j) {
      std::cout << array[i][j] << "  ";
      if (j == (nbcols - 1)) std::cout << std::endl;
    }
  }
}

int main() {
  // declare a vector on the host
  int **ptr_cpu = NULL, *ptr_gpu = NULL;
  const int nbcols = 4, nbrows = 5;
  int N = nbrows * nbcols;
  int nbytes = N * sizeof(int);
  ptr_cpu = new int*[nbrows];
  // for (int i = 0; i < nbrows; i++) ptr_cpu[i] = new int[nbcols];

  // !The memory should be contiguous on the host
  ptr_cpu[0] = (int*)malloc(nbytes);
  // ptr_cpu[0]=new int[N];
  for (int i = 1; i < nbrows; ++i) ptr_cpu[i] = ptr_cpu[i - 1] + nbcols;

  int k = 0;
  for (int i = 0; i < nbrows; ++i) {
    for (int j = 0; j < nbcols; ++j) ptr_cpu[i][j] = k++;
  };
  print_array(ptr_cpu, nbrows, nbcols);

  // allocate gpu memory
  hipMalloc(&ptr_gpu, nbytes);

  // copy data to gpu
  hipMemcpy(ptr_gpu, ptr_cpu[0], nbytes, hipMemcpyHostToDevice);
  if (ptr_gpu == NULL) {
    printf("Couldn't allocate GPU memory\n");
  };

  // 2d block: threads in x and y directions
  dim3 bs = dim3(nbrows, nbcols);
  // 1d grid: block in x direction
  int grid = 1;  // Only 1 block
  // grid and block dimensions
  int grid_dim = 1, block_dim = 2;
  gpu_array_swap<<<grid, bs, nbytes>>>(ptr_gpu, nbrows, nbcols, grid_dim,
                                       block_dim);

  //! Copy data from device to host
  hipDeviceSynchronize();
  memset(ptr_cpu, 0, nbytes);
  hipMemcpy(ptr_cpu[0], ptr_gpu, nbytes, hipMemcpyDeviceToHost);
  if (hipMemcpy(ptr_cpu[0], ptr_gpu, nbytes, hipMemcpyDeviceToHost) !=
      hipSuccess)
    std::cout << "Fail to copy back to cpu!" << std::endl;

  print_array(ptr_cpu, nbrows, nbcols);
  hipFree(ptr_gpu);
  free(ptr_cpu);
}
