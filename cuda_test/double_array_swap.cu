#include <iostream>
#include <algorithm>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cassert>
//define the chunk sizes that each threadblock will work on
#define BLKXSIZE 32
#define BLKYSIZE 4
#define BLKZSIZE 4
#define Q 19
#define lx 10
#define ly 10
#define lz 5

// for cuda error checking
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            return 1; \
        } \
    } while (0)

template <typename T> __device__ void swap ( T& a, T& b )
{
  T c(a); a=b; b=c;
}


__global__ void gpu_array_swap(int ptr_gpu[][ly][lz][Q]) {
  // int thread_id = thread_idx(grid_dim, block_dim);
  unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned idy = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned idz = blockIdx.z * blockDim.z + threadIdx.z;
  if ((idx < lx) && (idy < ly) && (idz < lz)) {
    for (size_t i = 1; i <= 9; i++)
          swap(ptr_gpu[idx][idy][idz][i], ptr_gpu[idx][idy][idz][i + 9]);
  }
}

void set_array(int array[][ly][lz][Q]) {
  int m = 0;
  for (int l = 0; l < Q; ++l) {
    for (int i = 0; i < lz; ++i) {
      for (int j = 0; j < ly; ++j) {
        for (int k = 0; k < lx; ++k) {
          array[i][j][k][l] = ++m;
        }
      }
    }
  }
}

void print_array(int array[][ly][lz][Q]) {
  for (int i = 0; i < lx; ++i) {
    for (int j = 0; j < ly; ++j) {
      for (int k = 0; k < lz; ++k) {
        for (int l = 0; l < Q; ++l) {
          std::cout << array[i][j][k][l] << "  ";
          if (l == (Q - 1)) std::cout << std::endl;
        }
      }
    }
  }
}

int main() {
  typedef int array_3d[ly][lz];
  typedef int array_4d[ly][lz][Q];
  const dim3 blockSize(BLKXSIZE, BLKYSIZE, BLKZSIZE);
  const dim3 gridSize(((lx + BLKXSIZE - 1) / BLKXSIZE),
                      ((ly + BLKYSIZE - 1) / BLKYSIZE),
                      ((lz + BLKZSIZE - 1) / BLKZSIZE));

  // pointers for data set storage via malloc
  array_4d* c;    // storage for result stored on host
  array_4d* d_c;  // storage for result computed on device
                  // allocate storage for data set
  if ((c = (array_4d*)malloc((lx * ly * lz * Q) * sizeof(int))) == 0) {
    fprintf(stderr, "malloc1 Fail \n");
    return 1;
  }
  set_array(c);
  print_array(c);
  
  // allocate GPU device buffers
  hipMalloc((void**)&d_c, (lx * ly * lz * Q) * sizeof(int));
  cudaCheckErrors("Failed to allocate device buffer");
hipMemcpy(d_c, c, ((lx * ly * lz * Q) * sizeof(int)),
             hipMemcpyHostToDevice);
  
  // compute result
  gpu_array_swap<<<gridSize, blockSize>>>(d_c);
  cudaCheckErrors("Kernel launch failure");
  // copy output data back to host

  hipMemcpy(c, d_c, ((lx * ly * lz * Q) * sizeof(int)),
             hipMemcpyDeviceToHost);
  cudaCheckErrors("CUDA memcpy failure");
  free(c);
  hipFree(d_c);
  cudaCheckErrors("hipFree fail");
  return 0;
}
