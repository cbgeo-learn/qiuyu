#include "hip/hip_runtime.h"
#include <iostream>
#include "vector_summation.cuh"
#include <algorithm>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <stdio.h>

GpuVector::GpuVector(int* vec_cpu,int N):N_(N)
{
  //! Allocate GPU mem
  int nbytes=N_*sizeof(int);
  hipMallocManaged((void **)&vec_gpu,nbytes);
  hipMallocManaged((void **)&vec_sum_device,sizeof(int));
  hipMemcpy(vec_gpu, vec_cpu, nbytes, hipMemcpyHostToDevice);
  if(vec_gpu == NULL || vec_sum_device == NULL )
  {printf("couldn't allocate GPU memory\n");}
}

void GpuVector::sum()
{
  //! 1D block
  int bs = 256;
  //! 2D grid 
  int s = ceil(sqrt((N_/sizeof(int) + bs - 1.) / bs));
  dim3 grid = dim3(s, s);

  //! Call kernel function from the host
  vector_sum_kernel<<<grid, bs>>>(vec_gpu,N_,vec_sum_device);

  //! Copy data from device to host
  hipDeviceSynchronize();
  hipMemcpy(&vec_sum_host, vec_sum_device, sizeof(int), hipMemcpyDeviceToHost);
}

GpuVector::~GpuVector(){hipFree(vec_gpu);}

int main()
{
  //! Declare a vector on the host
  int* vec_cpu;
  int N=100;
  int vec_cpu_sum=0;

  //! Allocate CPU mem
  int nbytes = N * sizeof(int);
  vec_cpu = (int *) malloc(nbytes);
  for (int i = 0; i < N; ++i)vec_cpu[i]=10;
  
  //! Declare a constructor
  GpuVector vec_gpu(vec_cpu,N);

  //! Calculate the sum on gpu
  std::cout<<"before running on gpu, sum="<<vec_gpu.vec_sum_host<<std::endl;
  vec_gpu.sum();

  //! Check if data is copied from gpu to cpu
  hipDeviceSynchronize();
  std::cout<<"After running on gpu, sum="<<vec_gpu.vec_sum_host<<std::endl;

  //! Calculate the sum on cpu
  std::cout<<"Before running on cpu, sum="<<vec_cpu_sum<<std::endl;
  hipDeviceSynchronize();
  for( size_t i = 0 ; i < N ; i++ )
  {vec_cpu_sum+=vec_cpu[i];}
  std::cout<<"After running on cpu, sum="<<vec_cpu_sum<<std::endl;
  
  free(vec_cpu); 
}

