#include <iostream>
#include <algorithm>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cassert>
//define the chunk sizes that each threadblock will work on
#define BLKXSIZE 32
#define BLKYSIZE 4
#define BLKZSIZE 4
#define Q 19
#define lx 10
#define ly 10
#define lz 5

// for cuda error checking
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            return 1; \
        } \
    } while (0)

template <typename T> __device__ void swap ( T& a, T& b )
{
  T c(a); a=b; b=c;
}

template <typename Ttype>
Ttype**** create_4d_harray() {
  Ttype**** array;
  array = new Ttype***[lx];
  assert(array != nullptr);
  
  array[0] = (Ttype***)malloc(sizeof(Ttype)*lx*ly*lz*Q);
  for (int i = 1; i < lx; i++) {//from i=0 to i=1;
    // array[i] = new Ttype**[ly];
    array[i] = array[i-1] + ly*lz*Q;
    assert(array[i] != nullptr);
    array[i][0] = (Ttype**)malloc(sizeof(Ttype)*ly*lz*Q);
    for (int j = 1; j < ly; j++) {
      //array[i][j] = new Ttype*[lz];
      array[i][j] = array[i][j-1] + lz*Q;
      assert(array[i][j] != nullptr);
      array[i][j][0] = (Ttype*)malloc(sizeof(Ttype)*Q);
      for (int k = 1; k < lz; k++) {
        //array[i][j][k] = new Ttype[n];  
        array[i][j][k] = array[i][j][k-1]+ Q;
        assert(array[i][j][k] != nullptr);
      }
    }
  }
  return array;
}

template <typename Ttype>
void free_4d_harray(Ttype**** array) {
  for (size_t i = 0; i < lx; ++i) {
    for (size_t j = 0; j < ly; ++j) {
      for (size_t k = 0; k < lz; ++k) {
        delete[] array[i][j][k];
      }
      delete[] array[i][j];
    }
    delete[] array[i];
  }
  delete[] array;
}


template <typename Ttype>
Ttype (*create_4d_darray())[ly][lz][Q] {
  // const int d_lx = lx, d_ly = ly, d_lz = lz, d_n = n;
  //Ttype array_4d[lx][ly][lz][Q];
  //typedef Ttype array_4d[ly][lz][Q];
  //array_4d* dptr_4d_array;
  Ttype(*dptr_4d_array)[ly][lz][Q];
  hipMalloc((void**)&dptr_4d_array,
             (lx * ly * lz * Q) * sizeof(Ttype));
  return dptr_4d_array;
}

__global__ void gpu_array_swap(int ptr_gpu[][ly][lz][Q]) {
  // int thread_id = thread_idx(grid_dim, block_dim);
  unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned idy = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned idz = blockIdx.z * blockDim.z + threadIdx.z;
  if ((idx < lx) && (idy < ly) && (idz < lz)) {
    for (size_t i = 1; i <= 9; i++)
          swap(ptr_gpu[idx][idy][idz][i], ptr_gpu[idx][idy][idz][i + 9]);
  }
}

void set_array(int**** array) {
  int m = 0;
  for (int l = 0; l < Q; ++l) {
    for (int i = 0; i < lz; ++i) {
      for (int j = 0; j < ly; ++j) {
        for (int k = 0; k < lx; ++k) {
          array[i][j][k][l] = ++m;
        }
      }
    }
  }
}

void print_array(int**** array) {
  for (int i = 0; i < lx; ++i) {
    for (int j = 0; j < ly; ++j) {
      for (int k = 0; k < lz; ++k) {
        for (int l = 0; l < Q; ++l) {
          std::cout << array[i][j][k][l] << "  ";
          if (l == (Q - 1)) std::cout << std::endl;
        }
      }
    }
  }
}

int main() {

  //int lx=10,ly=10,lz=5,Q=19;
  int**** host_4d_array=create_4d_harray<int>();
  int(* device_4d_array)[ly][lz][Q]=create_4d_darray<int>();
  
  const dim3 blockSize(BLKXSIZE, BLKYSIZE, BLKZSIZE);
  const dim3 gridSize(((lx + BLKXSIZE - 1) / BLKXSIZE),
                      ((ly + BLKYSIZE - 1) / BLKYSIZE),
                      ((lz + BLKZSIZE - 1) / BLKZSIZE));

  set_array(host_4d_array);
  print_array(host_4d_array);
  
  // allocate GPU device buffers

  cudaCheckErrors("Failed to allocate device buffer");
  hipMemcpy(device_4d_array, host_4d_array, (lx * ly * lz * Q) * sizeof(int),
             hipMemcpyHostToDevice);
  
  // compute result
  gpu_array_swap<<<gridSize, blockSize>>>(device_4d_array);
  cudaCheckErrors("Kernel launch failure");
  // copy output data back to host

  hipMemcpy(host_4d_array, device_4d_array, ((lx * ly * lz * Q) * sizeof(int)),
             hipMemcpyDeviceToHost);
  cudaCheckErrors("CUDA memcpy failure");
  free_4d_harray(host_4d_array);
  hipFree(device_4d_array);
  cudaCheckErrors("hipFree fail");
  return 0;
}
