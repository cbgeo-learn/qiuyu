#include <iostream>
#include "vector_summation.cuh"
#include <algorithm>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

GpuVector::GpuVector(int* vec_cpu,int nbytes){
  /* allocate GPU mem */    
  hipMallocManaged(&vec_gpu,nbytes);
  hipMemcpy(vec_gpu, vec_cpu, nbytes, hipMemcpyHostToDevice);
}

void GpuVector::sum()
{vector_sum_kernel<<<1, 1>>>(vec_gpu,length_,vec_sum);}

GpuVector::~GpuVector(){hipFree(vec_gpu);}

int main()
{ /*Declare a vector on the host*/
  int* vec_cpu;
  int N=100;
  int nbytes = N * sizeof(int);
  vec_cpu = (int *) malloc(nbytes);
  for (int i = 0; i < N; ++i)vec_cpu[i]=10;
  
  hipDeviceSynchronize();
  
  // Run kernel on 1M elements on the GPU
  GpuVector vec_gpu(vec_cpu,nbytes);
  
  std::cout<<vec_gpu.vec_sum<<std::endl;
  vec_gpu.sum();
  std::cout<<vec_gpu.vec_sum<<std::endl;
  
  hipDeviceSynchronize();
  int vec_cpu_sum=0;
  for( size_t i = 0 ; i < N ; i++ )
  {vec_cpu_sum+=vec_cpu[i];}
  std::cout<<vec_cpu_sum<<std::endl;
  
  free(vec_cpu); 
}

